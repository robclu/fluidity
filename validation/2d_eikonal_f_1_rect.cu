#include "hip/hip_runtime.h"
//==--- fluidity/validation/2d_eikonal_f_1_rect.cu --------- -*- C++ -*- ---==//
//            
//                                Fluidity
// 
//                      Copyright (c) 2019 Rob Clucas.
//
//  This file is distributed under the MIT License. See LICENSE for details.
//
//==------------------------------------------------------------------------==//
//
/// \file  2d_eikonal_f_1_rect.cu
/// \brief This file defines a two dimensional validation case for the Eikonal
///        solver where the speed function has speed f=1, and the source node
///        is places in the centre of the domain and is a rectangle.
//
//==------------------------------------------------------------------------==//

#include <fluidity/algorithm/fill.hpp>
#include <fluidity/container/device_tensor.hpp>
#include <fluidity/container/vec.hpp>
#include <fluidity/geometry/rectangle.hpp>
#include <fluidity/scheme/eikonal/fast_iterative.hpp>
#include <fluidity/solver/eikonal_solver.hpp>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <memory>

using namespace fluid;

using real_t = double;
using vec2_t = Vec2<real_t>;
using pos_t  = Vec2<real_t>;

static constexpr auto size_x = int{2000};
static constexpr auto size_y = int{2000};
static constexpr auto dims   = 2;
static constexpr auto res    = 1.0;

template <typename I, typename T, typename O>
auto output_data(I&& it, T d, O& output) -> void {
  if (d == 0) {
    for (auto i : range(it.size(d))) {
      output
        << std::setw(6)
        << std::setprecision(4)
        << std::scientific
        << *it.offset(i, d)
        << " ";
    }
    output << "\n";
  } else {
    for (auto i : range(it.size(d))) {
      output_data(it, d - 1, output);
      it.shift(1, d);
    }
    std::cout << "\n";
  }
}

template <typename I, typename T>
void write_data(I& it, T d) {
  std::ofstream output_file;
  auto filename = "box_2d_output.dat";
  output_file.open(filename, std::fstream::trunc);
  output_data(std::forward<I>(it), d, output_file);
  output_file.close();
}

template <typename I, typename T>
auto print_data(I&& it, T d) -> void {
  if (d == 0) {
    for (auto i : range(it.size(d))) {
      std::cout
        << std::setprecision(2)
        << std::scientific
        << *it.offset(i, d)
        << " ";
    }
    std::cout << "\n";
  } else {
    for (auto i : range(it.size(d))) {
      print_data(it, d - 1);
      it.shift(1, d);
    }
    std::cout << "\n";
  }
}

int main(int argc, char** argv) {
  // What we would ideally do (if this was optimized), is to create a context
  // with a device type for the system (this should be able to be determined
  // quite easily), which is either GPU, CPU, or defauly (which would choose
  // the optimal one).
  // auto context = Context::default();
  
  // TODO: Finish this example with the ideal interface ...

  // The test is going to be run on the device, so we use a device tensor.
  using storage_t = DeviceTensor<real_t, 2>;
  auto input = storage_t{size_x, size_y};

  // Fill the input data, we set each cell as the signed distance from the
  // center of the domain. Since everything is outside of the center cell, the
  // signed distance for all cells is positive.
  fill(input.multi_iterator(), [&] fluidity_host_device (auto& cell)
  {
    constexpr auto center_x = static_cast<real_t>(size_x) / 2.0;
    constexpr auto center_y = static_cast<real_t>(size_y) / 2.0;
    constexpr auto length_x = static_cast<real_t>(size_x) / 2.0;
    constexpr auto length_y = static_cast<real_t>(size_y) / 2.0;

    using namespace geometry;
    auto p = pos_t(flattened_id(dim_x), flattened_id(dim_y));

    *cell = Rect<real_t>{
      vec2_t(length_x, length_y), pos_t{center_x, center_y}
    }.distance(p);
  });

  // Create the output data from the input data. We don't care about the data
  // in the output tensor, so we just copy the metadata.
  auto output = input.copy_without_data();

  // Since this test uses a constant speed function, we do not need to create
  // speed data for the solver, and can just solve for the input data.
  solver::eikonal(input, output, res, scheme::eikonal::FastIterative());

  auto host_out    = output.as_host();
  auto host_out_it = host_out.multi_iterator();
  using iter_t     = std::decay_t<decltype(host_out_it)>;

  // Print the results ...
  auto outer_dim = iter_t::dimensions - 1;
  write_data(host_out_it, iter_t::dimensions - 1);
}
