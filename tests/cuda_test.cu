#include "hip/hip_runtime.h"
//==--- fluidity/test/cuda_test.cu-------------------------- -*- C++ -*- ---==//
//            
//                      Copyright (c) 2018 Rob Clucas.
//
//  This file is distributed under the MIT License. See LICENSE for details.
//
//==------------------------------------------------------------------------==//
//
/// \file  cuda_test.cu
/// \brief This file defines a simple test case to check that cuda works.
//
//==------------------------------------------------------------------------==//

#include <fluidity/utility/debug.hpp>
#include <stdlib.h>
#include <stdio.h>

__global__ void invoke_kernel(float sum, int iterations) {
  for (auto i = 0; i < iterations; ++i)
    sum += 1.0f;

  auto thread = threadIdx.x + blockIdx.x * blockDim.x;
  printf("Thread %02u, value %04.4f\n", thread, sum);
}

int main(int argc, char** argv) {
  int threads = 16, iterations = 1;
  float sum = 0.0f;

  if (argc > 1)
    threads = atoi(argv[1]);
  if (argc > 2)
    iterations = atoi(argv[2]);

  invoke_kernel<<<1, threads>>>(sum, iterations);
  fluidity_check_cuda_result(hipDeviceSynchronize());
}
