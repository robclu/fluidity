#include "hip/hip_runtime.h"
//==--- fluidity/validation/2d_eikonal_f_1.cu -------------- -*- C++ -*- ---==//
//            
//                                Fluidity
// 
//                      Copyright (c) 2019 Rob Clucas.
//
//  This file is distributed under the MIT License. See LICENSE for details.
//
//==------------------------------------------------------------------------==//
//
/// \file  2d_eikonal_f_1_triangle.cu
/// \brief This file defines a two dimensional validation case for the Eikonal
///        solver where the speed function has speed f=1, and the initial data
///        is the signed distance function to a triangle.
///
//==------------------------------------------------------------------------==//

#include <fluidity/algorithm/fill.hpp>
#include <fluidity/container/device_tensor.hpp>
#include <fluidity/geometry/triangle.hpp>
#include <fluidity/scheme/eikonal/fast_iterative.hpp>
#include <fluidity/solver/eikonal_solver.hpp>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <memory>

using namespace fluid;

using real_t = double;
using pos2_t = geometry::Pos2<real_t>;

static constexpr auto size_x = int{2000};
static constexpr auto size_y = int{2000};
static constexpr auto dims   = 2;
static constexpr auto res    = 1.0;

template <typename I, typename T, typename O>
auto output_data(I&& it, T d, O& output) -> void {
  if (d == 0) {
    for (auto i : range(it.size(d))) {
      output
        << std::setw(6)
        << std::setprecision(4)
        << std::scientific
        << *it.offset(i, d)
        << " ";
    }
    output << "\n";
  } else {
    for (auto i : range(it.size(d))) {
      output_data(it, d - 1, output);
      it.shift(1, d);
    }
    std::cout << "\n";
  }
}

template <typename I, typename T>
void write_data(I& it, T d) {
  std::ofstream output_file;
  auto filename = "triangle_2d_output.dat";
  output_file.open(filename, std::fstream::trunc);
  output_data(std::forward<I>(it), d, output_file);
  output_file.close();
}

template <typename I, typename T>
auto print_data(I&& it, T d) -> void {
  if (d == 0) {
    for (auto i : range(it.size(d))) {
      std::cout
        << std::setprecision(2)
        << std::scientific
        << *it.offset(i, d)
        << " ";
    }
    std::cout << "\n";
  } else {
    for (auto i : range(it.size(d))) {
      print_data(it, d - 1);
      it.shift(1, d);
    }
    std::cout << "\n";
  }
}

int main(int argc, char** argv) {
  // What we would ideally do (if this was optimized), is to create a context
  // with a device type for the system (this should be able to be determined
  // quite easily), which is either GPU, CPU, or defauly (which would choose
  // the optimal one).
  // auto context = Context::default();
  
  // TODO: Finish this example with the ideal interface ...

  // The test is going to be run on the device, so we use a device tensor.
  using storage_t = DeviceTensor<real_t, 2>;
  auto input = storage_t{size_x, size_y};

  // Fill the input data, we set each cell as the signed distance from the
  // center of the domain. Since everything is outside of the center cell, the
  // signed distance for all cells is positive.
  fill(input.multi_iterator(), [&] fluidity_host_device (auto& cell) {
    using namespace geometry;

    auto scaled_x = [&] (auto in) { return in * size_x; };
    auto scaled_y = [&] (auto in) { return in * size_y; };

    auto p0 = pos2_t{scaled_x(0.25), scaled_y(0.25)};
    auto p1 = pos2_t{scaled_x(0.75), scaled_y(0.25)};
    auto p2 = pos2_t{scaled_x(0.50), scaled_y(0.683)};
    auto  p = pos2_t(flattened_id(dim_x), flattened_id(dim_y));

    *cell = Triangle<real_t>(p0, p1, p2).distance(p);
  });

  // Create the output data from the input data. We don't care about the data
  // in the output tensor, so we just copy the metadata.
  auto output = input.copy_without_data();

  // Since this test uses a constant speed function, we do not need to create
  // speed data for the solver, and can just solve for the input data.
  solver::eikonal(input, output, res, scheme::eikonal::FastIterative());

  auto host_out    = output.as_host();
  auto host_out_it = host_out.multi_iterator();
  using iter_t     = std::decay_t<decltype(host_out_it)>;

  // Print the results ...
  auto outer_dim = iter_t::dimensions - 1;
  write_data(host_out_it, iter_t::dimensions - 1);
}
