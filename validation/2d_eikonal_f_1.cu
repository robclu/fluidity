#include "hip/hip_runtime.h"
//==--- fluidity/validation/2d_eikonal_f_1.cu -------------- -*- C++ -*- ---==//
//            
//                                Fluidity
// 
//                      Copyright (c) 2019 Rob Clucas.
//
//  This file is distributed under the MIT License. See LICENSE for details.
//
//==------------------------------------------------------------------------==//
//
/// \file  2d_eikonal_f_1.cu
/// \brief This file defines a two dimensional validation case for the Eikonal
///        solver where the speed function has speed f=1, and the source node
///        is places in the centre of the domain.
//
//==------------------------------------------------------------------------==//

#include <fluidity/algorithm/fill.hpp>
#include <fluidity/container/device_tensor.hpp>
#include <fluidity/geometry/sphere.hpp>
#include <fluidity/scheme/eikonal/fast_iterative.hpp>
#include <fluidity/solver/eikonal_solver.hpp>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <memory>

using namespace fluid;

using real_t = double;

static constexpr auto size_x   = int{1500};
static constexpr auto size_y   = int{1500};
static constexpr auto center_x = static_cast<real_t>(size_x) / 4.0;
static constexpr auto center_y = static_cast<real_t>(size_y) / 4.0;
static constexpr auto radius   = static_cast<real_t>(size_x) / 10.0;
static constexpr auto dims     = 2;
static constexpr auto res      = 1.0;

template <typename I, typename T, typename O>
auto output_data(I&& it, T d, O& output) -> void {
  if (d == 0) {
    for (auto i : range(it.size(d))) {
      output
        << std::setw(6)
        << std::setprecision(4)
        << std::scientific
        << *it.offset(i, d)
        << " ";
    }
    output << "\n";
  } else {
    for (auto i : range(it.size(d))) {
      output_data(it, d - 1, output);
      it.shift(1, d);
    }
    std::cout << "\n";
  }
}

template <typename I, typename T>
void write_data(I& it, T d) {
  std::ofstream output_file;
  auto filename = "output.dat";
  output_file.open(filename, std::fstream::trunc);
  output_data(std::forward<I>(it), d, output_file);
  output_file.close();
}

template <typename I, typename T>
auto print_data(I&& it, T d) -> void {
  if (d == 0) {
    for (auto i : range(it.size(d))) {
      std::cout
        << std::setprecision(2)
        << std::scientific
        << *it.offset(i, d)
        << " ";
    }
    std::cout << "\n";
  } else {
    for (auto i : range(it.size(d))) {
      print_data(it, d - 1);
      it.shift(1, d);
    }
    std::cout << "\n";
  }
}

int main(int argc, char** argv) {
  // What we would ideally do (if this was optimized), is to create a context
  // with a device type for the system (this should be able to be determined
  // quite easily), which is either GPU, CPU, or defauly (which would choose
  // the optimal one).
  // auto context = Context::default();
  
  // TODO: Finish this example with the ideal interface ...

  // The test is going to be run on the device, so we use a device tensor.
  using storage_t = DeviceTensor<real_t, 2>;
  auto input = storage_t{size_x, size_y};

  // Fill the input data, we set each cell as the signed distance from the
  // center of the domain. Since everything is outside of the center cell, the
  // signed distance for all cells is positive.
  fill(input.multi_iterator(), [&] fluidity_host_device (auto& cell)
  {
    using namespace geometry;
    auto p = Pos<real_t>{
      flattened_id(dim_x), flattened_id(dim_y), flattened_id(dim_z)
    };
    *cell = Sphere<real_t>(center_x, center_y, 0.0, radius).distance(p);
  });

  // Create the output data from the input data. We don't care about the data
  // in the output tensor, so we just copy the metadata.
  auto output = input.copy_without_data();

  // Since this test uses a constant speed function, we do not need to create
  // speed data for the solver, and can just solve for the input data.
  solver::eikonal(input, output, res, scheme::eikonal::FastIterative());

  auto host_out    = output.as_host();
  auto host_out_it = host_out.multi_iterator();
  using iter_t     = std::decay_t<decltype(host_out_it)>;

  // Print the results ...
  auto outer_dim = iter_t::dimensions - 1;
  write_data(host_out_it, iter_t::dimensions - 1);
}
